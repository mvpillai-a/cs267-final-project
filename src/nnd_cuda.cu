#include "hip/hip_runtime.h"
// nnd_cuda.cu - Real CUDA implementation
#include "nnd_cuda.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

namespace nndescent_cuda {

// Constants
const char OLD = '0';
const char NEW = '1';
const int NONE = -1;

// Squared Euclidean distance on GPU
__device__ float squared_euclidean_cuda(
    const float* data,
    int idx0,
    int idx1,
    int dim
) {
    float result = 0.0f;
    for (int i = 0; i < dim; ++i) {
        float diff = data[idx0 * dim + i] - data[idx1 * dim + i];
        result += diff * diff;
    }
    return result;
}

// GPU heap operations
__device__ int device_checked_push(
    int* indices,
    float* keys,
    char* flags,
    int heap_start,
    int heap_size,
    int idx,
    float key,
    char flag
) {
    if (key >= keys[heap_start]) {
        return 0;
    }
    
    // Check if already in heap
    for (int i = 0; i < heap_size; ++i) {
        if (indices[heap_start + i] == idx) {
            return 0;
        }
    }
    
    // Siftdown
    int current = 0;
    while (true) {
        int left_child = 2 * current + 1;
        int right_child = left_child + 1;
        int swap;
        
        if (left_child >= heap_size) {
            break;
        } else if (right_child >= heap_size) {
            swap = (keys[heap_start + left_child] > key) ? left_child : -1;
        } else if (keys[heap_start + left_child] >= keys[heap_start + right_child]) {
            swap = (keys[heap_start + left_child] > key) ? left_child : -1;
        } else {
            swap = (keys[heap_start + right_child] > key) ? right_child : -1;
        }
        
        if (swap == -1) break;
        
        indices[heap_start + current] = indices[heap_start + swap];
        keys[heap_start + current] = keys[heap_start + swap];
        flags[heap_start + current] = flags[heap_start + swap];
        current = swap;
    }
    
    indices[heap_start + current] = idx;
    keys[heap_start + current] = key;
    flags[heap_start + current] = flag;
    return 1;
}

// Initialize with random neighbors
__global__ void init_random_kernel(
    int* indices,
    float* keys,
    char* flags,
    const float* data,
    int num_points,
    int n_neighbors,
    int dim,
    unsigned int seed
) {
    int idx0 = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx0 < num_points) {
        hiprandState state;
        hiprand_init(seed, idx0, 0, &state);
        
        int heap_start = idx0 * n_neighbors;
        
        // Initialize heap with random neighbors
        for (int j = 0; j < n_neighbors; ++j) {
            int idx1 = hiprand(&state) % num_points;
            float d = squared_euclidean_cuda(data, idx0, idx1, dim);
            device_checked_push(indices, keys, flags, heap_start, n_neighbors, idx1, d, NEW);
        }
    }
}

// Sample candidates kernel
__global__ void sample_candidates_kernel(
    const int* indices,
    const char* flags,
    int* new_candidates,
    int* old_candidates,
    int* new_counts,
    int* old_counts,
    int num_points,
    int n_neighbors,
    int max_candidates,
    unsigned int seed
) {
    int idx0 = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx0 < num_points) {
        hiprandState state;
        hiprand_init(seed, idx0, 0, &state);
        
        int new_count = 0;
        int old_count = 0;
        
        // Sample from current neighbors
        for (int j = 0; j < n_neighbors; ++j) {
            int idx1 = indices[idx0 * n_neighbors + j];
            if (idx1 == NONE) continue;
            
            char flag = flags[idx0 * n_neighbors + j];
            int priority = hiprand(&state);
            
            if (flag == NEW && new_count < max_candidates) {
                new_candidates[idx0 * max_candidates + new_count] = idx1;
                new_count++;
            } else if (flag == OLD && old_count < max_candidates) {
                old_candidates[idx0 * max_candidates + old_count] = idx1;
                old_count++;
            }
        }
        
        new_counts[idx0] = new_count;
        old_counts[idx0] = old_count;
    }
}

// Generate updates kernel
__global__ void generate_updates_kernel(
    const float* data,
    const int* indices,
    const float* keys,
    const int* new_candidates,
    const int* old_candidates,
    const int* new_counts,
    const int* old_counts,
    int* updates,
    float* update_distances,
    int* update_count,
    int num_points,
    int n_neighbors,
    int max_candidates,
    int dim,
    int max_updates_per_point
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < num_points) {
        int count = 0;
        int update_offset = i * max_updates_per_point;
        
        int new_count = new_counts[i];
        int old_count = old_counts[i];
        
        // New-new pairs
        for (int j = 0; j < new_count; ++j) {
            int idx0 = new_candidates[i * max_candidates + j];
            for (int k = j + 1; k < new_count; ++k) {
                int idx1 = new_candidates[i * max_candidates + k];
                
                float d = squared_euclidean_cuda(data, idx0, idx1, dim);
                
                if (d < keys[idx0 * n_neighbors] || d < keys[idx1 * n_neighbors]) {
                    if (count < max_updates_per_point) {
                        updates[update_offset + count * 2] = idx0;
                        updates[update_offset + count * 2 + 1] = idx1;
                        update_distances[update_offset + count] = d;
                        count++;
                    }
                }
            }
        }
        
        // New-old pairs
        for (int j = 0; j < new_count; ++j) {
            int idx0 = new_candidates[i * max_candidates + j];
            for (int k = 0; k < old_count; ++k) {
                int idx1 = old_candidates[i * max_candidates + k];
                
                float d = squared_euclidean_cuda(data, idx0, idx1, dim);
                
                if (d < keys[idx0 * n_neighbors] || d < keys[idx1 * n_neighbors]) {
                    if (count < max_updates_per_point) {
                        updates[update_offset + count * 2] = idx0;
                        updates[update_offset + count * 2 + 1] = idx1;
                        update_distances[update_offset + count] = d;
                        count++;
                    }
                }
            }
        }
        
        update_count[i] = count;
    }
}

// Apply updates kernel
__global__ void apply_updates_kernel(
    int* indices,
    float* keys,
    char* flags,
    const int* updates,
    const float* update_distances,
    const int* update_counts,
    int num_points,
    int n_neighbors,
    int max_updates_per_point
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_points) {
        // Process all updates for this point
        for (int i = 0; i < num_points; ++i) {
            int update_offset = i * max_updates_per_point;
            int count = update_counts[i];
            
            for (int j = 0; j < count; ++j) {
                int idx0 = updates[update_offset + j * 2];
                int idx1 = updates[update_offset + j * 2 + 1];
                float d = update_distances[update_offset + j];
                
                if (idx0 == idx) {
                    int heap_start = idx0 * n_neighbors;
                    device_checked_push(indices, keys, flags, heap_start, n_neighbors, idx1, d, NEW);
                }
                if (idx1 == idx) {
                    int heap_start = idx1 * n_neighbors;
                    device_checked_push(indices, keys, flags, heap_start, n_neighbors, idx0, d, NEW);
                }
            }
        }
    }
}

// Main CUDA NN-Descent function
void nn_descent_cuda(
    const nndescent::Matrix<float>& data,
    nndescent::HeapList<float>& current_graph,
    int n_neighbors,
    nndescent::RandomState& rng_state,
    int max_candidates,
    int n_iters,
    float delta,
    int n_threads,
    bool verbose,
    const std::string& metric
) {
    int num_points = data.nrows();
    int dim = data.ncols();
    
    if (max_candidates == NONE) {
        max_candidates = std::min(60, n_neighbors);
    }
    
    // Allocate device memory
    float* d_data;
    int* d_indices;
    float* d_keys;
    char* d_flags;
    int* d_new_candidates;
    int* d_old_candidates;
    int* d_new_counts;
    int* d_old_counts;
    int* d_updates;
    float* d_update_distances;
    int* d_update_counts;
    
    hipMalloc(&d_data, num_points * dim * sizeof(float));
    hipMalloc(&d_indices, num_points * n_neighbors * sizeof(int));
    hipMalloc(&d_keys, num_points * n_neighbors * sizeof(float));
    hipMalloc(&d_flags, num_points * n_neighbors * sizeof(char));
    hipMalloc(&d_new_candidates, num_points * max_candidates * sizeof(int));
    hipMalloc(&d_old_candidates, num_points * max_candidates * sizeof(int));
    hipMalloc(&d_new_counts, num_points * sizeof(int));
    hipMalloc(&d_old_counts, num_points * sizeof(int));
    
    int max_updates_per_point = max_candidates * max_candidates;
    hipMalloc(&d_updates, num_points * max_updates_per_point * 2 * sizeof(int));
    hipMalloc(&d_update_distances, num_points * max_updates_per_point * sizeof(float));
    hipMalloc(&d_update_counts, num_points * sizeof(int));
    
    // Initialize arrays
    hipMemcpy(d_data, data.m_ptr, num_points * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_indices, 0xFF, num_points * n_neighbors * sizeof(int));  // Set to -1
    hipMemset(d_keys, 0x7F, num_points * n_neighbors * sizeof(float));   // Set to large value
    hipMemset(d_flags, NEW, num_points * n_neighbors * sizeof(char));
    
    // Launch configuration
    int block_size = 256;
    int grid_size = (num_points + block_size - 1) / block_size;
    
    if (verbose) {
        std::cout << "CUDA NN descent for " << n_iters << " iterations" << std::endl;
    }
    
    // Initialize with random neighbors
    init_random_kernel<<<grid_size, block_size>>>(
        d_indices, d_keys, d_flags, d_data, num_points, n_neighbors, dim, rng_state[0]
    );
    hipDeviceSynchronize();
    
    // Main NN-Descent loop
    for (int iter = 0; iter < n_iters; ++iter) {
        if (verbose) {
            std::cout << "\t" << iter + 1 << "  /  " << n_iters << std::endl;
        }
        
        // Sample candidates
        sample_candidates_kernel<<<grid_size, block_size>>>(
            d_indices, d_flags, d_new_candidates, d_old_candidates,
            d_new_counts, d_old_counts, num_points, n_neighbors, max_candidates,
            rng_state[0] + iter
        );
        hipDeviceSynchronize();
        
        // Generate updates
        generate_updates_kernel<<<grid_size, block_size>>>(
            d_data, d_indices, d_keys, d_new_candidates, d_old_candidates,
            d_new_counts, d_old_counts, d_updates, d_update_distances, d_update_counts,
            num_points, n_neighbors, max_candidates, dim, max_updates_per_point
        );
        hipDeviceSynchronize();
        
        // Apply updates
        apply_updates_kernel<<<grid_size, block_size>>>(
            d_indices, d_keys, d_flags, d_updates, d_update_distances, d_update_counts,
            num_points, n_neighbors, max_updates_per_point
        );
        hipDeviceSynchronize();
        
        // Count changes (for verbose output)
        if (verbose) {
            int h_update_counts[num_points];
            hipMemcpy(h_update_counts, d_update_counts, num_points * sizeof(int), hipMemcpyDeviceToHost);
            
            int total_updates = 0;
            for (int i = 0; i < num_points; ++i) {
                total_updates += h_update_counts[i];
            }
            
            std::cout << "\t\t" << total_updates << " updates generated" << std::endl;
        }
    }
    
    // Copy results back to host
    std::vector<int> h_indices(num_points * n_neighbors);
    std::vector<float> h_keys(num_points * n_neighbors);
    
    hipMemcpy(h_indices.data(), d_indices, num_points * n_neighbors * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_keys.data(), d_keys, num_points * n_neighbors * sizeof(float), hipMemcpyDeviceToHost);
    
    // Copy to HeapList and apply distance correction
    for (int i = 0; i < num_points; ++i) {
        for (int j = 0; j < n_neighbors; ++j) {
            current_graph.indices(i, j) = h_indices[i * n_neighbors + j];
            current_graph.keys(i, j) = std::sqrt(h_keys[i * n_neighbors + j]);
        }
    }
    
    // Sort the results
    current_graph.heapsort();
    
    // Clean up
    hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_keys);
    hipFree(d_flags);
    hipFree(d_new_candidates);
    hipFree(d_old_candidates);
    hipFree(d_new_counts);
    hipFree(d_old_counts);
    hipFree(d_updates);
    hipFree(d_update_distances);
    hipFree(d_update_counts);
}

} // namespace nndescent_cuda